#include "hip/hip_runtime.h"
/* compile with: nvcc -O3 hw1.cu -o hw1 */

// ================================================================================================
// includes and typedefs
// ================================================================================================
#include <stdio.h>
#include <sys/time.h>

#define IMG_DIMENSION 32
#define N_IMG_PAIRS 10000

typedef unsigned char uchar;
#define OUT

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

// ================================================================================================
// cpu functions
// ================================================================================================
double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

/* we won't load actual files. just fill the images with random bytes */
void load_image_pairs(uchar *images1, uchar *images2) {
    srand(0);
    for (int i = 0; i < N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION; i++) {
        images1[i] = rand() % 256;
        images2[i] = rand() % 256;
    }
}

__global__ bool is_in_image_bounds(int i, int j) {
    return (i >= 0) && (i < IMG_DIMENSION) && (j >= 0) && (j < IMG_DIMENSION);
}

__global__ uchar local_binary_pattern(uchar *image, int i, int j) {
    uchar center = image[i * IMG_DIMENSION + j];
    uchar pattern = 0;
    if (is_in_image_bounds(i - 1, j - 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j - 1)] >= center) << 7;
    if (is_in_image_bounds(i - 1, j    )) pattern |= (image[(i - 1) * IMG_DIMENSION + (j    )] >= center) << 6;
    if (is_in_image_bounds(i - 1, j + 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j + 1)] >= center) << 5;
    if (is_in_image_bounds(i    , j + 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j + 1)] >= center) << 4;
    if (is_in_image_bounds(i + 1, j + 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j + 1)] >= center) << 3;
    if (is_in_image_bounds(i + 1, j    )) pattern |= (image[(i + 1) * IMG_DIMENSION + (j    )] >= center) << 2;
    if (is_in_image_bounds(i + 1, j - 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j - 1)] >= center) << 1;
    if (is_in_image_bounds(i    , j - 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j - 1)] >= center) << 0;
    return pattern;
}


void image_to_histogram(uchar *image, int *histogram) {
    memset(histogram, 0, sizeof(int) * 256);
    for (int i = 0; i < IMG_DIMENSION; i++) {
        for (int j = 0; j < IMG_DIMENSION; j++) {
            uchar pattern = local_binary_pattern(image, i, j);
            histogram[pattern]++;
        }
    }
}

double histogram_distance(int *h1, int *h2) {
    /* we'll use the chi-square distance */
    double distance = 0;
    for (int i = 0; i < 256; i++) {
        if (h1[i] + h2[i] != 0) {
            distance += ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
        }
    }
    return distance;
}

// ================================================================================================
// __device__ functions and __global__ kernels
// ================================================================================================
__global__ void image_to_hisogram_simple(uchar *image1, OUT int *hist1) {
    // assuming single thread block
    int i = threadIdx.y;
    int j = threadIdx.x;

    if (i < IMG_DIMENSION && j < IMG_DIMENSION) {
        uchar pattern = local_binary_pattern(image1, i, j);
        // atomicAdd is used to avoid different threads accessing hist1[pattern] simultaneously
        atomicAdd_block(hist1[pattern], 1); 
    }

}
__global__ void histogram_distance(int *hist1, int *hist2, OUT double *distance) {
    // assuming single thread block
    int i = threadIdx.x;
    if (i < 256) {
        distance[i] = ((double)SQR(hist1[i] - hist2[i])) / (hist1[i] + hist2[i]);
    }

}

__global__ void kogge_stone_scan(float *A, int length) {
    int tid = threadIdx.x;
    int increment;
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid >= stride) {
            increment = A[tid - stride];
        }
        __syncthreads();

        if (tid >= stride) {
            A[tid] += increment;
        }
        __syncthreads();
    }
}

// ================================================================================================
// main
// ================================================================================================
int main() {
    uchar *images1; /* we concatenate all images in one huge array */
    uchar *images2;
    CUDA_CHECK( hipHostAlloc(&images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
    CUDA_CHECK( hipHostAlloc(&images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );

    load_image_pairs(images1, images2);
    double t_start, t_finish;
    double total_distance;

    /* using CPU */
    printf("\n=== CPU ===\n");
    int histogram1[256];
    int histogram2[256];
    t_start  = get_time_msec();
    for (int i = 0; i < N_IMG_PAIRS; i++) {
        image_to_histogram(&images1[i * IMG_DIMENSION * IMG_DIMENSION], histogram1);
        image_to_histogram(&images2[i * IMG_DIMENSION * IMG_DIMENSION], histogram2);
        total_distance += histogram_distance(histogram1, histogram2);
    }
    t_finish = get_time_msec();
    printf("average distance between images %f\n", total_distance / N_IMG_PAIRS);
    printf("total time %f [msec]\n", t_finish - t_start);

    /* using GPU task-serial */
    printf("\n=== GPU Task Serial ===\n");
    do { /* do {} while (0): to keep variables inside this block in their own scope. remove if you prefer otherwise */
        /* Your Code Here */
        uchar *gpu_image1, *gpu_image2;
        int *gpu_hist1, *gpu_hist2;
        double *gpu_hist_distance_vec; 

        // allocate on gpu global memory
        CUDA_CHECK( cudaAlloc((void**)&gpu_image1, IMG_DIMENSION * IMG_DIMENSION * sizeof(uchar)) );
        CUDA_CHECK( cudaAlloc((void**)&gpu_image2, IMG_DIMENSION * IMG_DIMENSION * sizeof(uchar)) );
        CUDA_CHECK( cudaAlloc((void**)&gpu_hist1, 256 * sizeof(int)) );
        CUDA_CHECK( cudaAlloc((void**)&gpu_hist2, 256 * sizeof(int)) );
        CUDA_CHECK( cudaAlloc((void**)&gpu_hist_distance_vec, 256 * sizeof(double)) );

        double cpu_hist_distance;

        t_start = get_time_msec();
        for (int i = 0; i < N_IMG_PAIRS; i++) {
            // copy relevant images from images1 and images2 to gpu_image1 and gpu_image2
            CUDA_CHECK( hipMemcpy(gpu_image1, &images1[i * IMG_DIMENSION * IMG_DIMENSION], hipMemcpyHostToDevice);
            CUDA_CHECK( hipMemcpy(gpu_image2, &images2[i * IMG_DIMENSION * IMG_DIMENSION], hipMemcpyHostToDevice);

            // using 32x32=1024 threads, calculate the binary pattern and historgram for each pixel
            dim3 dimBlock(IMG_DIMENSION, IMG_DIMENSION);
            image_to_hisogram_simple<<<1, dimBlock>>>(gpu_image1, gpu_hist1);
            image_to_hisogram_simple<<<1, dimBlock>>>(gpu_image2, gpu_hist2);

            // calculate distance
            histogram_distance<<<1, 256>>>(gpu_hist1, gpu_hist2, gpu_hist_distance_vec);
            kogge_stone_scan<<<1, 256>>>(gpu_hist_distance_vec, 256)

            // copy gpu_hist_distance_vec[255] to cpu_hist_distance 
            CUDA_CHECK( hipMemcpy((void*)&cpu_hist_distance, &gpu_hist_distance[255], hipMemcpyDeviceToHost);
            
            total_distance += cpu_hist_distance;
        }
        CUDA_CHECK(hipDeviceSynchronize());
        t_finish = get_time_msec();
        printf("average distance between images %f\n", total_distance / N_IMG_PAIRS);
        printf("total time %f [msec]\n", t_finish - t_start);
    } while (0);

    /* using GPU task-serial + images and histograms in shared memory */
    printf("\n=== GPU Task Serial with shared memory ===\n");
    /* Your Code Here */
    printf("average distance between images %f\n", total_distance / N_IMG_PAIRS);
    printf("total time %f [msec]\n", t_finish - t_start);

    /* using GPU + batching */
    printf("\n=== GPU Batching ===\n");
    /* Your Code Here */
    printf("average distance between images %f\n", total_distance / N_IMG_PAIRS);
    printf("total time %f [msec]\n", t_finish - t_start);

    // Free
    CUDA_CHECK( hipFree(gpu_image1) );
    CUDA_CHECK( hipFree(gpu_image2) );
    CUDA_CHECK( hipFree(gpu_hist1) );
    CUDA_CHECK( hipFree(gpu_hist2) );
    CUDA_CHECK( hipFree(gpu_hist_distance) );

    return 0;
}
